#include "hip/hip_runtime.h"
// for printing arrays
#include <iostream>
#include <iomanip>
#include <sstream>
#include <string>
// for min
#include <algorithm>

#include "cufixnum/util/cuda_wrap.h"
#include "cufixnum/array/fixnum_array.cuh"

namespace cuFIXNUM {

// TODO: The only device function in this file is the dispatch kernel
// mechanism, which could arguably be placed elsewhere, thereby
// allowing this file to be compiled completely for the host.

// Notes: Read programming guide Section K.3
// - Can prefetch unified memory
// - Can advise on location of unified memory

// TODO: Can I use smart pointers? unique_ptr?

// TODO: Clean this up
namespace {
    typedef std::uint8_t byte;

    template< typename T >
    static byte *as_byte_ptr(T *ptr) {
        return reinterpret_cast<byte *>(ptr);
    }

    template< typename T >
    static const byte *as_byte_ptr(const T *ptr) {
        return reinterpret_cast<const byte *>(ptr);
    }

    // TODO: refactor from word_fixnum.
    template< typename T >
    T ceilquo(T n, T d) {
        return (n + d - 1) / d;
    }
}

template< typename fixnum_t >
fixnum_array<fixnum_t> *
fixnum_array<fixnum_t>::create(size_t nelts) {
    fixnum_array *a = new fixnum_array;
    a->nelts = nelts;
    if (nelts > 0) {
        size_t nbytes = nelts * fixnum_t::BYTES;
        cuda_malloc_managed(&a->ptr, nbytes);
    }
    return a;
}

template< typename fixnum_t >
template< typename T >
fixnum_array<fixnum_t> *
fixnum_array<fixnum_t>::create(size_t nelts, T init) {
    fixnum_array *a = create(nelts);
    byte *p = as_byte_ptr(a->ptr);

    const byte *in = as_byte_ptr(&init);
    byte elt[fixnum_t::BYTES];
    memset(elt, 0, fixnum_t::BYTES);
    std::copy(in, in + sizeof(T), elt);

    for (uint32_t i = 0; i < nelts; ++i, p += fixnum_t::BYTES)
        fixnum_t::from_bytes(p, elt, fixnum_t::BYTES);
    return a;
}

template< typename fixnum_t >
fixnum_array<fixnum_t> *
fixnum_array<fixnum_t>::create(const byte *data, size_t total_bytes, size_t bytes_per_elt) {
    // FIXME: Should handle this error more appropriately
    if (total_bytes == 0 || bytes_per_elt == 0)
        return nullptr;

    size_t nelts = ceilquo(total_bytes, bytes_per_elt);
    fixnum_array *a = create(nelts);

    byte *p = as_byte_ptr(a->ptr);
    const byte *d = data;
    for (size_t i = 0; i < nelts; ++i) {
        fixnum_t::from_bytes(p, d, bytes_per_elt);
        p += fixnum_t::BYTES;
        d += bytes_per_elt;
    }
    return a;
}

// TODO: This doesn't belong here.
template< typename word_ft >
void
rotate_array(word_ft *out, const word_ft *in, int nelts, int words_per_elt, int i) {
    if (i < 0) {
        int j = -i;
        i += nelts * ceilquo(j, nelts);
        assert(i >= 0 && i < nelts);
        i = nelts - i;
    } else if (i >= nelts)
        i %= nelts;
    int pivot = i * words_per_elt;
    int nwords = nelts * words_per_elt;
    std::copy(in, in + nwords - pivot, out + pivot);
    std::copy(in + nwords - pivot, in + nwords, out);
}


// TODO: Find a way to return a wrapper that just modifies the requested indices
// on the fly, rather than copying the whole array. Hard part will be making it
// work with map/dispatch.
template< typename fixnum_t >
fixnum_array<fixnum_t> *
fixnum_array<fixnum_t>::rotate(int i) {
    fixnum_array *a = create(length());
    byte *p = as_byte_ptr(a->ptr);
    const byte *q = as_byte_ptr(ptr);
    rotate_array(p, q, nelts, fixnum_t::BYTES, i);
    return a;
}

template< typename fixnum_t >
fixnum_array<fixnum_t> *
fixnum_array<fixnum_t>::repeat(int ntimes) {
    fixnum_array *a = create(length() * ntimes);
    byte *p = as_byte_ptr(a->ptr);
    const byte *q = as_byte_ptr(ptr);
    int nbytes = nelts * fixnum_t::BYTES;
    for (int i = 0; i < ntimes; ++i, p += nbytes)
        std::copy(q, q + nbytes, p);
    return a;
}

template< typename fixnum_t >
fixnum_array<fixnum_t> *
fixnum_array<fixnum_t>::rotations(int ntimes) {
    fixnum_array *a = create(nelts * ntimes);
    byte *p = as_byte_ptr(a->ptr);
    const byte *q = as_byte_ptr(ptr);
    int nbytes = nelts * fixnum_t::BYTES;
    for (int i = 0; i < ntimes; ++i, p += nbytes)
        rotate_array(p, q, nelts, fixnum_t::BYTES, i);
    return a;
}


template< typename fixnum_t >
int
fixnum_array<fixnum_t>::set(int idx, const byte *data, size_t nbytes) {
    // FIXME: Better error handling
    if (idx < 0 || idx >= nelts)
        return -1;

    int off = idx * fixnum_t::BYTES;
    const byte *q = as_byte_ptr(ptr);
    return fixnum_t::from_bytes(q + off, data, nbytes);
}

template< typename fixnum_t >
fixnum_array<fixnum_t>::~fixnum_array() {
    if (nelts > 0)
        cuda_free(ptr);
}

template< typename fixnum_t >
int
fixnum_array<fixnum_t>::length() const {
    return nelts;
}

template< typename fixnum_t >
size_t
fixnum_array<fixnum_t>::retrieve_into(byte *dest, size_t dest_space, int idx) const {
    if (idx < 0 || idx > nelts) {
        // FIXME: This is not the right way to handle an "index out of
        // bounds" error.
        return 0;
    }
    const byte *q = as_byte_ptr(ptr);
    return fixnum_t::to_bytes(dest, dest_space, q + idx * fixnum_t::BYTES);
}

// FIXME: Can return fewer than nelts elements.
template< typename fixnum_t >
void
fixnum_array<fixnum_t>::retrieve_all(byte *dest, size_t dest_space, int *dest_nelts) const {
    const byte *p = as_byte_ptr(ptr);
    byte *d = dest;
    int max_dest_nelts = dest_space / fixnum_t::BYTES;
    *dest_nelts = std::min(nelts, max_dest_nelts);
    for (int i = 0; i < *dest_nelts; ++i) {
        fixnum_t::to_bytes(d, fixnum_t::BYTES, p);
        p += fixnum_t::BYTES;
        d += fixnum_t::BYTES;
    }
}

namespace {
    std::string
    fixnum_as_str(const uint8_t *fn, int nbytes) {
        std::ostringstream ss;

        for (int i = nbytes - 1; i >= 0; --i) {
            // These IO manipulators are forgotten after each use;
            // i.e. they don't apply to the next output operation (whether
            // it be in the next loop iteration or in the conditional
            // below.
            ss << std::setfill('0') << std::setw(2) << std::hex;
            ss << static_cast<int>(fn[i]);
            if (i && !(i & 3))
                ss << ' ';
        }
        return ss.str();
    }
}

template< typename fixnum_t >
std::ostream &
operator<<(std::ostream &os, const fixnum_array<fixnum_t> *fn_arr) {
    constexpr int fn_bytes = fixnum_t::BYTES;
    constexpr size_t bufsz = 4096;
    uint8_t arr[bufsz];
    int nelts;

    fn_arr->retrieve_all(arr, bufsz, &nelts);
    os << "( ";
    if (nelts < fn_arr->length()) {
        os << "insufficient space to retrieve array";
    } else if (nelts > 0) {
        os << fixnum_as_str(arr, fn_bytes);
        for (int i = 1; i < nelts; ++i)
            os << ", " << fixnum_as_str(arr + i*fn_bytes, fn_bytes);
    }
    os << " )" << std::flush;
    return os;
}


template< template <typename> class Func, typename fixnum_t, typename... Args >
__global__ void
dispatch(int nelts, Args... args) {
    // Get the slot index for the current thread.
    int blk_tid_offset = blockDim.x * blockIdx.x;
    int tid_in_blk = threadIdx.x;
    int idx = (blk_tid_offset + tid_in_blk) / fixnum_t::SLOT_WIDTH;

    if (idx < nelts) {
        // TODO: Find a way to load each argument into a register before passing
        // it to fn, and then unpack the return values where they belong. This
        // will guarantee that all operations happen on registers, rather than
        // inadvertently operating on memory.

        Func<fixnum_t> fn;
        // TODO: This offset calculation is entwined with fixnum layout and so
        // belongs somewhere else.
        int off = idx * fixnum_t::layout::WIDTH + fixnum_t::layout::laneIdx();
        // TODO: This is hiding a sin against memory aliasing / management /
        // type-safety.
        fn(args[off]...);
    }
}

template< typename fixnum_t >
template< template <typename> class Func, typename... Args >
void
fixnum_array<fixnum_t>::map(Args... args) {
    // TODO.opt this can be made faster by tuning carefully
    int block_size = cuda_get_cores_per_sm();

    // FIXME: WARPSIZE should come from slot_layout
    // constexpr int WARPSIZE = 32;

    int nelts = std::min( { args->length()... } );
    int fixnums_per_block = block_size / fixnum_t::SLOT_WIDTH;

    // FIXME: nblocks could be too big for a single kernel call to handle
    int nblocks = ceilquo(nelts, fixnums_per_block);

    // nblocks > 0 iff nelts > 0
    if (nblocks > 0) {
        hipStream_t stream;
        cuda_check(hipStreamCreate(&stream));
//         cuda_stream_attach_mem(stream, src->ptr);
//         cuda_stream_attach_mem(stream, ptr);
        cuda_check(hipStreamSynchronize(stream));

        dispatch<Func, fixnum_t ><<< nblocks, block_size, 0, stream >>>(nelts, args->ptr...);

        cuda_check(hipPeekAtLastError());
        cuda_check(hipStreamSynchronize(stream));
        cuda_check(hipStreamDestroy(stream));

        // FIXME: Only synchronize when retrieving data from array
        cuda_device_synchronize();
    }
}

} // End namespace cuFIXNUM
